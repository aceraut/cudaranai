#include "hip/hip_runtime.h"
// This file implements multiple Initializer classes, meant to initialize
// parameters in a neural network.
//
// Further notes on these initalizers can be found here:
// https://pytorch.org/docs/stable/nn.init.html

#include "common.cuh"
#include "initializer.cuh"

#include <chrono>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>

namespace nnv2 {

// Kernel to initialize data with normal distribution
__global__ void normal_init_kernel(
    int size,
    float *vec,
    float mean,
    float stddev,
    unsigned seed) {
  CUDA_GRID_STRIDE_LOOP(idx, size) {
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    vec[idx] = mean + stddev * hiprand_normal(&state);
  }
}

static void normal_init(VecType &vec, float s) {
  float *vec_raw = RAW_PTR(vec);
  unsigned seed =
      (unsigned)std::chrono::steady_clock::now().time_since_epoch().count();

  int size = vec.size();
  int grid_size = utils::div_ceil(size, BLOCK_SIZE);

  normal_init_kernel<<<grid_size, BLOCK_SIZE>>>(size, vec_raw, 0, s, seed);
  CUDA_POST_KERNEL_CHECK;
}

// Kernel to initialize data with uniform distribution
__global__ void
uniform_init_kernel(int size, float *vec, float a, float b, unsigned seed) {
  CUDA_GRID_STRIDE_LOOP(idx, size) {
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    vec[idx] = a + (b - a) * hiprand_uniform(&state);
  }
}

static void uniform_init(VecType &vec, float r) {
  float *vec_raw = RAW_PTR(vec);
  unsigned seed =
      (unsigned)std::chrono::steady_clock::now().time_since_epoch().count();

  int size = vec.size();
  int grid_size = utils::div_ceil(size, BLOCK_SIZE);

  uniform_init_kernel<<<grid_size, BLOCK_SIZE>>>(size, vec_raw, -r, r, seed);
  CUDA_POST_KERNEL_CHECK;
}

void LecunNormal::initialize(Array *a, int fan_in, int fan_out) const {
  float s = sqrtf(1.0 / fan_in);
  normal_init(a->get_vec(), s);
}

void XavierNormal::initialize(Array *a, int fan_in, int fan_out) const {
  float s = sqrtf(2.0 / (fan_in + fan_out));
  normal_init(a->get_vec(), s);
}

void KaimingNormal::initialize(Array *a, int fan_in, int fan_out) const {
  float s = sqrtf(2.0 / fan_in);
  normal_init(a->get_vec(), s);
}

void SimplyNormal::initialize(Array *a, int fan_in, int fan_out) const {
  normal_init(a->get_vec(), 0.1);
}

void LecunUniform::initialize(Array *a, int fan_in, int fan_out) const {
  float r = sqrtf(1.0 / fan_in);
  uniform_init(a->get_vec(), r);
}

void XavierUniform::initialize(Array *a, int fan_in, int fan_out) const {
  float r = sqrtf(6.0 / (fan_in + fan_out));
  uniform_init(a->get_vec(), r);
}

void KaimingUniform::initialize(Array *a, int fan_in, int fan_out) const {
  float r = sqrtf(6.0 / fan_in);
  uniform_init(a->get_vec(), r);
}

void SimplyUniform::initialize(Array *a, int fan_in, int fan_out) const {
  uniform_init(a->get_vec(), 0.01);
}

} // namespace nnv2