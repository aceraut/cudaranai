#include "hip/hip_runtime.h"
#include "common.cuh"

#include <cmath>
#include <functional>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

namespace nnv2 {

__global__ void matmul_kernel(float *output, const float *input1,
                              const float *input2, int m, int n, int k,
                              int broadcast) {
    __shared__ float input1_tile[TILE_DIM][TILE_DIM];
    __shared__ float input2_tile[TILE_DIM][TILE_DIM];

    // Calculate offsets of the matrices
    int batch_idx = blockIdx.z;
    if (broadcast != 1) {
        input1 += batch_idx * m * k;
    }
    if (broadcast != 2) {
        input2 += batch_idx * k * n;
    }
    output += batch_idx * m * n;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_DIM + ty;
    int col = bx * TILE_DIM + tx;

    // Loop over input tiles to calculate the dot value
    float value = 0;
    int tile_count = (k + TILE_DIM - 1) / TILE_DIM;

    for (int i = 0; i < tile_count; i++) {
        // Load input tiles to shared memory
        if (row < m && i * TILE_DIM + tx < k) {
            input1_tile[ty][tx] = input1[row * k + i * TILE_DIM + tx];
        } else {
            input1_tile[ty][tx] = 0;
        }
        if (col < n && i * TILE_DIM + ty < k) {
            input2_tile[ty][tx] = input2[(i * TILE_DIM + ty) * n + col];
        } else {
            input2_tile[ty][tx] = 0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_DIM; j++) {
            value += input1_tile[ty][j] * input2_tile[j][tx];
        }
        __syncthreads();
    }

    if (row < m && col < n) {
        output[row * n + col] = value;
    }
}

// Another implementation (which is really slow and not as precise)
__global__ void matmul_kernel_exp(float *output, const float *input1,
                                  const float *input2, int m, int n, int k,
                                  int broadcast) {
    __shared__ float block1[BM * BK];
    __shared__ float block2[BK * BN];

    float thread_output[TM * TN] = {0.0};
    float tile1[TM] = {0.0};
    float tile2[TM] = {0.0};

    // Calculate offsets of the matrices
    const int batch_idx = blockIdx.z;
    if (broadcast != 1) {
        input1 += batch_idx * m * k;
    }
    if (broadcast != 2) {
        input2 += batch_idx * k * n;
    }
    output += batch_idx * m * n;

    // Indices of current block in output matrix
    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Indices of current chunk in said block
    const int thread_row = threadIdx.x / (BN / TN);
    const int thread_col = threadIdx.x % (BN / TN);

    const int block_nthreads = blockDim.x;
    const int chunk1_stride = block_nthreads / BK;
    const int chunk2_stride = block_nthreads / BN;

    const int chunk1_row = threadIdx.x / BK;
    const int chunk1_col = threadIdx.x % BK;
    const int chunk2_row = threadIdx.x / BN;
    const int chunk2_col = threadIdx.x % BN;

    for (int block_st = 0; block_st < k; block_st += BK) {
        for (int offset = 0; offset < BM; offset += chunk1_stride) {
            const int row_block1 = offset + chunk1_row;
            const int col_block1 = chunk1_col;
            const int row_input1 = block_row * BM + row_block1;
            const int col_input1 = block_st + col_block1;

            block1[row_block1 * BK + col_block1] =
                (row_input1 < m && col_input1 < k)
                    ? input1[row_input1 * k + col_input1]
                    : 0;
        }

        for (int offset = 0; offset < BK; offset += chunk2_stride) {
            const int row_block2 = offset + chunk2_row;
            const int col_block2 = chunk2_col;
            const int row_input2 = block_st + row_block2;
            const int col_input2 = block_col * BN + col_block2;

            block2[row_block2 * BN + col_block2] =
                (row_input2 < k && col_input2 < n)
                    ? input2[row_input2 * n + col_input2]
                    : 0;
        }
        __syncthreads();

        for (int dot_idx = 0; dot_idx < BK; dot_idx++) {
            for (int i = 0; i < TM; i++) {
                tile1[i] = block1[(thread_row * TM + i) * BK + dot_idx];
            }
            for (int i = 0; i < TN; i++) {
                tile2[i] = block2[dot_idx * BN + thread_col * TN + i];
            }
            for (int x = 0; x < TM; x++) {
                for (int y = 0; y < TN; y++) {
                    thread_output[x * TN + y] += tile1[x] * tile2[y];
                }
            }
        }
        __syncthreads();
    }

    for (int x = 0; x < TM; x++) {
        for (int y = 0; y < TN; y++) {
            const int row_output = block_row * BM + thread_row * TM + x;
            const int col_output = block_col * BN + thread_col * TN + y;
            if (row_output < m && col_output < n) {
                output[row_output * n + col_output] +=
                    thread_output[x * TN + y];
            }
        }
    }
}

__global__ void transpose_kernel(float *output, const float *input, int m,
                                 int n) {
    __shared__ float input_tile[TILE_DIM][TILE_DIM];

    int batch_idx = blockIdx.z;
    input += batch_idx * m * n;
    output += batch_idx * n * m;

    int bx = blockIdx.y;
    int by = blockIdx.x;
    int tx = threadIdx.y;
    int ty = threadIdx.x;

    int row = bx * TILE_DIM + tx;
    int col = by * TILE_DIM + ty;

    if (row < m && col < n) {
        input_tile[tx][ty] = input[row * n + col];
        __syncthreads();
        output[col * m + row] = input_tile[tx][ty];
    }
}

__global__ void sum_kernel(int size, float *output, const float *input,
                           int axis_size, int stride) {
    CUDA_GRID_STRIDE_LOOP(idx, size) {
        int base = (idx / stride) * axis_size * stride + (idx % stride);

        float value = 0;
        for (int i = 0; i < axis_size; i++) {
            value += input[base + i * stride];
        }
        output[idx] = value;
    }
}

__global__ void mean_kernel(int size, float *output, const float *input,
                            int axis_size, int stride) {
    CUDA_GRID_STRIDE_LOOP(idx, size) {
        int base = (idx / stride) * axis_size * stride + (idx % stride);

        float value = 0;
        for (int i = 0; i < axis_size; i++) {
            value += input[base + i * stride];
        }
        output[idx] = value / axis_size;
    }
}

namespace ops {

void add(Array *output, const Array *input1, const Array *input2) {
    const VecType &input1_vec = input1->get_vec();
    const VecType &input2_vec = input2->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(input1_vec.size(), input2_vec.size(),
             "ops::add: size mismatch between inputs");
    CHECK_EQ(output_vec.size(), input1_vec.size(),
             "ops::add: size mismatched between input and output");

    thrust::transform(input1_vec.begin(), input1_vec.end(), input2_vec.begin(),
                      output_vec.begin(), thrust::plus<float>());
}

void add(Array *output, const Array *input, float value) {
    const VecType &input_vec = input->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(output_vec.size(), input_vec.size(),
             "ops::add: size mismatch between input and output");

    thrust::transform(input_vec.begin(), input_vec.end(), output_vec.begin(),
                      [value] __device__(float x) { return x + value; });
}

void subtract(Array *output, const Array *input1, const Array *input2) {
    const VecType &input1_vec = input1->get_vec();
    const VecType &input2_vec = input2->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(input1_vec.size(), input2_vec.size(),
             "ops::subtract: size mismatch between inputs");
    CHECK_EQ(output_vec.size(), input1_vec.size(),
             "ops::subtract: size mismatch between input and outputs");

    thrust::transform(input1_vec.begin(), input1_vec.end(), input2_vec.begin(),
                      output_vec.begin(), thrust::minus<float>());
}

void subtract(Array *output, const Array *input, float value) {
    const VecType &input_vec = input->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(output_vec.size(), input_vec.size(),
             "ops::subtract: size mismatch between input and output");

    thrust::transform(input_vec.begin(), input_vec.end(), output_vec.begin(),
                      [value] __device__(float x) { return x - value; });
}

void multiply(Array *output, const Array *input1, const Array *input2) {
    const VecType &input1_vec = input1->get_vec();
    const VecType &input2_vec = input2->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(input1_vec.size(), input2_vec.size(),
             "ops::multiply: size mismatch between inputs");
    CHECK_EQ(output_vec.size(), input1_vec.size(),
             "ops::multiply: size mismatch between input and outputs");

    thrust::transform(input1_vec.begin(), input1_vec.end(), input2_vec.begin(),
                      output_vec.begin(), thrust::multiplies<float>());
}

void multiply(Array *output, const Array *input, float value) {
    const VecType &input_vec = input->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(output_vec.size(), input_vec.size(),
             "ops::multiply: size mismatch between input and output");

    thrust::transform(input_vec.begin(), input_vec.end(), output_vec.begin(),
                      [value] __device__(float x) { return x * value; });
}

void divide(Array *output, const Array *input1, const Array *input2) {
    const VecType &input1_vec = input1->get_vec();
    const VecType &input2_vec = input2->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(input1_vec.size(), input2_vec.size(),
             "ops::divide: size mismatch between inputs");
    CHECK_EQ(output_vec.size(), input1_vec.size(),
             "ops::divide: size mismatch between input and outputs");

    thrust::transform(input1_vec.begin(), input1_vec.end(), input2_vec.begin(),
                      output_vec.begin(), thrust::divides<float>());
}

void log(Array *output, const Array *input) {
    const VecType &input_vec = input->get_vec();
    VecType &output_vec = output->get_vec();

    CHECK_EQ(output_vec.size(), input_vec.size(),
             "ops::log: size mismatch between input and output");

    thrust::transform(input_vec.begin(), input_vec.end(), output_vec.begin(),
                      [] __device__(float e) { return logf(e); });
}

// Performs matrix multiplication with two modes based on `broadcast` value:
//
// `broadcast == 0` (default):
// - For 2D inputs: single matrix multiplication.
// - For higher dimensions: batch matrix multiplication on corresponding
// matrices.
//
// `broadcast == 1 or 2`:
// - Batch matrix multiplication between a batch A and a single matrix B.
// - If `broadcast == 1`, B is the first input; if `broadcast == 2`, B is the
// second input.
void matmul(Array *output, const Array *input1, const Array *input2,
            int broadcast) {
    const ShapeType &output_shape = output->get_shape();
    const ShapeType &input1_shape = input1->get_shape();
    const ShapeType &input2_shape = input2->get_shape();

    CHECK_COND(input1_shape.size() > 1,
               "ops::matmul: shape error at first input");
    CHECK_COND(input2_shape.size() > 1,
               "ops::matmul: shape error at second input");
    CHECK_COND(output_shape.size() > 1, "ops::matmul: shape error at output");

    // Additional dimension check for broadcast case
    if (broadcast == 1) {
        CHECK_EQ(input1_shape.size(), 2,
                 "ops::matmul: shape error at first input");
    } else if (broadcast == 2) {
        CHECK_EQ(input2_shape.size(), 2,
                 "ops::matmul: shape error at second input");
    }

    // Calculate batch size and validate
    int batch_size =
        std::accumulate(output_shape.begin(), output_shape.end() - 2, 1,
                        std::multiplies<int>());
    int bs_input1 =
        std::accumulate(input1_shape.begin(), input1_shape.end() - 2, 1,
                        std::multiplies<int>());
    int bs_input2 =
        std::accumulate(input2_shape.begin(), input2_shape.end() - 2, 1,
                        std::multiplies<int>());

    if (broadcast != 1) {
        CHECK_EQ(batch_size, bs_input1, "ops::matmul: batch size mismatch");
    }
    if (broadcast != 2) {
        CHECK_EQ(batch_size, bs_input2, "ops::matmul: batch size mismatch");
    }

    // Validate matrix dimension
    int m = *(input1_shape.rbegin() + 1);
    int k = *(input1_shape.rbegin());
    int n = *(input2_shape.rbegin());
    int input2_h = *(input2_shape.rbegin() + 1);
    int output_h = *(output_shape.rbegin() + 1);
    int output_w = *(output_shape.rbegin());

    CHECK_EQ(k, input2_h, "ops::matmul: shape mismatch between inputs");
    CHECK_EQ(m, output_h,
             "ops::matmul: shape mismatch between first input and output");
    CHECK_EQ(n, output_w,
             "ops::matmul: shape mismatch between second input and output");

    // Launch kernels
    dim3 grid_dim(utils::quotient_ceil(n, TILE_DIM),
                  utils::quotient_ceil(m, TILE_DIM), batch_size);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    float *output_raw = RAW_PTR(output->get_vec());
    const float *input1_raw = RAW_PTR(input1->get_vec());
    const float *input2_raw = RAW_PTR(input2->get_vec());

    matmul_kernel<<<grid_dim, block_dim>>>(output_raw, input1_raw, input2_raw,
                                           m, n, k, broadcast);
    CUDA_POST_KERNEL_CHECK;

    // dim3 grid_dim(utils::quotient_ceil(n, BN), utils::quotient_ceil(m, BM),
    //               batch_size);
    // dim3 block_dim((BM * BN) / (TM * TN));

    // float *output_raw = RAW_PTR(output->get_vec());
    // const float *input1_raw = RAW_PTR(input1->get_vec());
    // const float *input2_raw = RAW_PTR(input2->get_vec());

    // matmul_kernel_exp<<<grid_dim, block_dim>>>(
    //     output_raw, input1_raw, input2_raw, m, n, k, broadcast);
    // CUDA_POST_KERNEL_CHECK;
}

// Performs matrix tranpose. If the input has more than 2 dimensions, batch
// matrix transpose is performed, which requires output to have the same batch
// size as the input array
void transpose(Array *output, const Array *input) {
    const ShapeType &output_shape = output->get_shape();
    const ShapeType &input_shape = input->get_shape();

    // Check if the dimensions are at least 2
    CHECK_COND(input_shape.size() > 1, "ops::transpose: shape error at input");
    CHECK_COND(output_shape.size() > 1,
               "ops::transpose: shape error at output");

    // Calculate batch size and validate
    int batch_size =
        std::accumulate(output_shape.begin(), output_shape.end() - 2, 1,
                        std::multiplies<int>());
    int bs_input = std::accumulate(input_shape.begin(), input_shape.end() - 2,
                                   1, std::multiplies<int>());
    CHECK_EQ(batch_size, bs_input, "ops::transpose: batch size mismatch");

    // Validate matrix dimension
    int m = *(input_shape.rbegin() + 1);
    int n = *(input_shape.rbegin());
    int output_h = *(output_shape.rbegin() + 1);
    int output_w = *(output_shape.rbegin());

    CHECK_EQ(m, output_w,
             "ops::transpose: shape mismatch between input and output");
    CHECK_EQ(n, output_h,
             "ops::transpose: shape mismatch between input and output");

    // Launch kernels
    dim3 grid_dim(utils::quotient_ceil(n, TILE_DIM),
                  utils::quotient_ceil(m, TILE_DIM), batch_size);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    float *output_raw = RAW_PTR(output->get_vec());
    const float *input_raw = RAW_PTR(input->get_vec());

    transpose_kernel<<<grid_dim, block_dim>>>(output_raw, input_raw, m, n);
    CUDA_POST_KERNEL_CHECK;
}

// Calculates sum of array elements along a given axis. The parameter `reduce`
// indicates whether the dimension at `axis` in input array is removed in the
// output.
void sum(Array *output, const Array *input, int axis, bool reduce) {
    const ShapeType &input_shape = input->get_shape();
    const ShapeType &output_shape = output->get_shape();

    CHECK_COND(axis >= 0,
               "ops::sum: support for negative axis isn't implemented");
    CHECK_COND(axis < input_shape.size(), "ops::sum: axis is out of bound");

    // Validate output shape
    // If `reduce` is true, remove the element at `axis` from output shape
    ShapeType reduced_shape = input->get_shape();
    if (reduce && input_shape.size() > 1) {
        reduced_shape.erase(reduced_shape.begin() + axis);
    } else {
        reduced_shape[axis] = 1;
    }
    CHECK_EQ(reduced_shape, output_shape, "ops::sum: shape error at output");

    // Launch kernels
    int output_size = output->get_vec().size();
    int axis_size = input_shape[axis];
    int stride = std::accumulate(input_shape.begin() + axis + 1,
                                 input_shape.end(), 1, std::multiplies<int>());
    int grid_size = utils::quotient_ceil(output_size, BLOCK_SIZE);

    float *output_raw = RAW_PTR(output->get_vec());
    const float *input_raw = RAW_PTR(input->get_vec());

    sum_kernel<<<grid_size, BLOCK_SIZE>>>(output_size, output_raw, input_raw,
                                          axis_size, stride);
    CUDA_POST_KERNEL_CHECK;
}

// Calculates mean value of array elements along a given axis. The parameter
// `reduce` indicates whether the dimension at `axis` in input array is removed
// in the output.
void mean(Array *output, const Array *input, int axis, bool reduce) {
    const ShapeType &input_shape = input->get_shape();
    const ShapeType &output_shape = output->get_shape();

    CHECK_COND(axis >= 0,
               "ops::mean: support for negative axis isn't implemented");
    CHECK_COND(axis < input_shape.size(), "ops::mean: axis is out of bound");

    // Validate output shape
    // If `reduce` is true, remove the element at `axis` from output shape
    ShapeType reduced_shape = input->get_shape();
    if (reduce && input_shape.size() > 1) {
        reduced_shape.erase(reduced_shape.begin() + axis);
    } else {
        reduced_shape[axis] = 1;
    }
    CHECK_EQ(reduced_shape, output_shape, "ops::mean: shape error at output");

    // Launch kernels
    int output_size = output->get_vec().size();
    int axis_size = input_shape[axis];
    int stride = std::accumulate(input_shape.begin() + axis + 1,
                                 input_shape.end(), 1, std::multiplies<int>());
    int grid_size = utils::quotient_ceil(output_size, BLOCK_SIZE);

    float *output_raw = RAW_PTR(output->get_vec());
    const float *input_raw = RAW_PTR(input->get_vec());

    mean_kernel<<<grid_size, BLOCK_SIZE>>>(output_size, output_raw, input_raw,
                                           axis_size, stride);
    CUDA_POST_KERNEL_CHECK;
}

} // namespace ops
} // namespace nnv2