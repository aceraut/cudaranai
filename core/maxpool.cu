#include "hip/hip_runtime.h"
#include "common.cuh"
#include "maxpool.cuh"

#include <cfloat>
#include <cmath>
#include <vector>

#include <hip/hip_runtime.h>

namespace nnv2 {

// The forward phase of max-pooling layer involves selecting the pixel with the
// largest value in every smaller local patches of the input feature maps and
// place them in the output feature maps.
__global__ void maxpool_forward_kernel(int size, float *output,
                                       int *max_indices, const float *input,
                                       int in_h, int in_w, int pad_h, int pad_w,
                                       int filter_h, int filter_w, int stride_h,
                                       int stride_w, int out_h, int out_w,
                                       int in_stride, int out_stride) {
    // Each thread handles a pixel in the output image
    CUDA_GRID_STRIDE_LOOP(idx, size) {
        // Point to input and output images that this thread handles
        int feat_idx = blockIdx.y;
        input += feat_idx * in_stride;
        output += feat_idx * out_stride;
        max_indices += feat_idx * out_stride;

        // Coord in output image
        int out_x = (idx / out_w) % out_h;
        int out_y = idx % out_w;

        // Locate the base coords of the section in the input image that
        // affects (out_x, out_y) of the output image
        int in_x_start = out_x * stride_h - pad_h;
        int in_y_start = out_y * stride_w - pad_w;
        int in_x_end = fminf(in_x_start + filter_h, in_h);
        int in_y_end = fminf(in_y_start + filter_w, in_w);
        in_x_start = fmaxf(in_x_start, 0);
        in_y_start = fmaxf(in_y_start, 0);

        // Loop over the local patch and select the pixel with largest value
        float max_val = -FLT_MAX;
        float max_idx = -1;

        for (int in_x = in_x_start; in_x < in_x_end; in_x++) {
            for (int in_y = in_y_start; in_y < in_y_end; in_y++) {
                int in_idx = in_x * in_w + in_y;
                if (input[in_idx] > max_val) {
                    max_idx = in_idx;
                    max_val = input[in_idx];
                }
            }
        }
        output[idx] = max_val;
        max_indices[idx] = max_idx;
    }
}

void maxpool_forward(Array *output, const Array *input,
                     thrust::device_vector<int> &indices, int pad_h, int pad_w,
                     int filter_h, int filter_w, int stride_h, int stride_w) {
    const ShapeType &output_shape = output->get_shape();
    const ShapeType &input_shape = input->get_shape();

    CHECK_EQ(output_shape.size(), 4, "maxpool_forward: output shape error");
    CHECK_EQ(input_shape.size(), 4, "maxpool_forward: input shape error");
    CHECK_EQ(indices.size(), output->get_vec().size(),
             "maxpool_forward: size mismatch beetween indices and output");

    int batch_size = input_shape[0];
    int in_feats = input_shape[1];

    CHECK_EQ(output_shape[0], batch_size, "maxpool_forward: batch size error");
    CHECK_EQ(output_shape[1], in_feats, "maxpool_forward: feature size error");

    int in_h = input_shape[2];
    int in_w = input_shape[3];
    int in_stride = in_h * in_w;
    int out_h = output_shape[2];
    int out_w = output_shape[3];
    int size = out_h * out_w; // is also out_stride
    dim3 grid_dim(utils::div_ceil(size, BLOCK_SIZE), batch_size * in_feats);

    float *output_raw = RAW_PTR(output->get_vec());
    int *indices_raw = RAW_PTR(indices);
    const float *input_raw = RAW_PTR(input->get_vec());

    maxpool_forward_kernel<<<grid_dim, BLOCK_SIZE>>>(
        size, output_raw, indices_raw, input_raw, in_h, in_w, pad_h, pad_w,
        filter_h, filter_w, stride_h, stride_w, out_h, out_w, in_stride, size);
    CUDA_POST_KERNEL_CHECK;
}

// Each output gradient is propagated back to its pooled position in the input.
__global__ void maxpool_backward_kernel(int size, float *input_grad,
                                        const float *output_grad,
                                        const int *max_indices, int in_stride,
                                        int out_stride) {
    CUDA_GRID_STRIDE_LOOP(idx, size) {
        int feat_idx = blockIdx.y;
        input_grad += feat_idx * in_stride;
        output_grad += feat_idx * out_stride;
        max_indices += feat_idx * out_stride;

        int pooled_index = max_indices[idx];
        float grad_value = output_grad[idx];

        atomicAdd(&input_grad[pooled_index], grad_value);
    }
}

void maxpool_backward(Array *input_grad, const Array *output_grad,
                      const thrust::device_vector<int> &indices, int pad_h,
                      int pad_w, int filter_h, int filter_w, int stride_h,
                      int stride_w) {
    const ShapeType &input_grad_shape = input_grad->get_shape();
    const ShapeType &output_grad_shape = output_grad->get_shape();

    CHECK_EQ(input_grad_shape.size(), 4,
             "maxpool_backward: input gradient shape error");
    CHECK_EQ(output_grad_shape.size(), 4,
             "maxpool_backward: output gradient shape error");
    CHECK_EQ(indices.size(), output_grad->get_vec().size(),
             "maxpool_backward: size mismatch between indices and output grad");

    int batch_size = input_grad_shape[0];
    int in_feats = input_grad_shape[1];

    CHECK_EQ(output_grad_shape[0], batch_size,
             "maxpool_backward: batch size error");
    CHECK_EQ(output_grad_shape[1], in_feats,
             "maxpool_backward: feature size error");

    int in_stride = input_grad_shape[2] * input_grad_shape[3];
    int out_stride = output_grad_shape[2] * output_grad_shape[3];
    dim3 grid_dim(utils::div_ceil(out_stride, BLOCK_SIZE),
                  batch_size * in_feats);

    float *input_grad_raw = RAW_PTR(input_grad->get_vec());
    const float *output_grad_raw = RAW_PTR(output_grad->get_vec());
    const int *indices_raw = RAW_PTR(indices);

    maxpool_backward_kernel<<<grid_dim, BLOCK_SIZE>>>(
        out_stride, input_grad_raw, output_grad_raw, indices_raw, in_stride,
        out_stride);
    CUDA_POST_KERNEL_CHECK;
}

MaxPool2D::MaxPool2D(int pad_h, int pad_w, int kernel_h, int kernel_w,
                     int stride_h, int stride_w)
    : pad_h(pad_h), pad_w(pad_w), kernel_h(kernel_h), kernel_w(kernel_w),
      stride_h(stride_h), stride_w(stride_w) {}

void MaxPool2D::forward() {
    const Array *input = prev->get_output();
    const ShapeType &input_shape = input->get_shape();

    int batch_size = input_shape[0];
    int in_feats = input_shape[1];
    int in_h = input_shape[2];
    int in_w = input_shape[3];
    int out_h = (in_h + 2 * pad_h - kernel_h) / stride_h + 1;
    int out_w = (in_w + 2 * pad_w - kernel_w) / stride_w + 1;

    utils::set_array_ptr(output, {batch_size, in_feats, out_h, out_w});
    indices.resize(output->get_vec().size());

    maxpool_forward(output.get(), input, indices, pad_h, pad_w, kernel_h,
                    kernel_w, stride_h, stride_w);
}

void MaxPool2D::backward() {
    const Array *input = prev->get_output();
    const Array *output_grad = next->get_grad();

    utils::set_array_ptr(grad, input->get_shape());

    maxpool_backward(grad.get(), output_grad, indices, pad_h, pad_w, kernel_h,
                     kernel_w, stride_h, stride_w);
}

} // namespace nnv2