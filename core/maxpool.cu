#include "hip/hip_runtime.h"
// This implements the MaxPool2D layer, a variant of pooling layer.
//
// This layer is typically used after the convolution layer and is used to
// reduce the spatial dimensions of the input data, by taking pixels with the
// maximum value in smaller grids of the input image to the output image.

#include "common.cuh"
#include "maxpool.cuh"

#include <cfloat>
#include <cmath>
#include <vector>

#include <hip/hip_runtime.h>

namespace nnv2 {

__global__ void maxpool_forward_kernel(int size, float *output,
                                       float *max_indices, const float *input,
                                       int in_h, int in_w, int pad_h, int pad_w,
                                       int filter_h, int filter_w, int stride_h,
                                       int stride_w, int out_h, int out_w,
                                       int in_stride, int out_stride) {
    // each thread handles a pixel in the output image
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
         idx += blockIdx.x * blockDim.x) {
        int out_x = (idx / out_w) % out_h;
        int out_y = idx % out_w;
        int feat_idx = idx / out_w / out_h;

        input += feat_idx * in_stride;
        output += feat_idx * out_stride;
        max_indices += feat_idx * out_stride;

        int in_x_start = out_x * stride_h - pad_h;
        int in_y_start = out_y * stride_w - pad_w;
        int in_x_end = fminf(in_x_start + filter_h, in_h);
        int in_y_end = fminf(in_y_start + filter_w, in_w);
        in_x_start = fmaxf(in_x_start, 0);
        in_y_start = fmaxf(in_y_start, 0);

        float max_val = -FLT_MAX;
        float max_idx = -1;
        for (int in_x = in_x_start; in_x < in_x_end; in_x++) {
            for (int in_y = in_y_start; in_y < in_y_end; in_y++) {
                int in_idx = in_x * in_w + in_y;
                if (input[in_idx] > max_val) {
                    max_idx = in_idx;
                    max_val = input[in_idx];
                }
            }
        }

        output[idx] = max_val;
        max_indices[idx] = max_idx;
    }
}

void maxpool_forward(Array *output, const Array *input, Array *indices,
                     int pad_h, int pad_w, int filter_h, int filter_w,
                     int stride_h, int stride_w) {
    CHECK_EQ(output->get_shape().size(), 4,
             "maxpool_forward: output shape error");
    CHECK_EQ(input->get_shape().size(), 4,
             "maxpool_forward: input shape error");

    int batch_size = input->get_shape()[0];
    int in_feats = input->get_shape()[1];

    CHECK_EQ(output->get_shape()[0], batch_size,
             "maxpool_forward: batch size error");
    CHECK_EQ(output->get_shape()[1], in_feats,
             "maxpool_forward: batch size error");
    CHECK_EQ(indices->get_vec().size(), output->get_vec().size(),
             "maxpool_forward: size mismatch beetween indices and output");

    int in_h = input->get_shape()[2];
    int in_w = input->get_shape()[3];
    int in_stride = in_h * in_w;
    int out_h = output->get_shape()[2];
    int out_w = output->get_shape()[3];
    int out_stride = out_h * out_w;

    float *output_raw = RAW_PTR(output->get_vec());
    float *indices_raw = RAW_PTR(indices->get_vec());
    const float *input_raw = RAW_PTR(input->get_vec());

    int size = batch_size * in_feats * out_h * out_w;
    int grid_size = ceil((float)size / BLOCK_SIZE);

    maxpool_forward_kernel<<<grid_size, BLOCK_SIZE>>>(
        size, output_raw, indices_raw, input_raw, in_h, in_w, pad_h, pad_w,
        filter_h, filter_w, stride_h, stride_w, out_h, out_w, in_stride,
        out_stride);
    CUDA_POST_KERNEL_CHECK;
}

__global__ void
maxpool_backward_kernel(int size, float *input_grad, const float *output_grad,
                        const float *max_indices, int in_h, int in_w, int pad_h,
                        int pad_w, int filter_h, int filter_w, int stride_h,
                        int stride_w, int out_h, int out_w, int in_stride,
                        int out_stride) {
    // each thread handles a pixel in the input image
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
         idx += blockIdx.x * blockDim.x) {
        // coord in input image
        int in_x = (idx / in_w) % in_h + pad_h;
        int in_y = idx % in_w + pad_w;
        int feat_idx = idx / out_w / out_h;

        input_grad += feat_idx * in_stride;
        output_grad += feat_idx * out_stride;
        max_indices += feat_idx * out_stride;

        // locate the base coords of the section in the output image that
        // depends on pixel (in_x, in_y) of the input image
        int out_x_start =
            (in_x < filter_h) ? 0 : (in_x - filter_h) / stride_h + 1;
        int out_x_end = fminf(out_h, in_x / stride_h + 1);
        int out_y_start =
            (in_y < filter_w) ? 0 : (in_y - filter_w) / stride_w + 1;
        int out_y_end = fminf(out_w, in_y / stride_w + 1);

        float value = 0;
        in_x -= pad_h;
        in_y -= pad_w; // since max_indices are unpadded

        for (int out_x = out_x_start; out_x < out_x_end; out_x++) {
            for (int out_y = out_y_start; out_y < out_y_end; out_y++) {
                if (max_indices[out_x * out_w + out_y] == in_x * in_w + in_y) {
                    value += output_grad[out_x * out_w + out_y];
                }
            }
        }
        input_grad[idx] = value;
    }
}

void maxpool_backward(Array *input_grad, const Array *output_grad,
                      const Array *indices, int pad_h, int pad_w, int filter_h,
                      int filter_w, int stride_h, int stride_w) {
    CHECK_EQ(input_grad->get_shape().size(), 4,
             "maxpool_backward: input gradient shape error");
    CHECK_EQ(output_grad->get_shape().size(), 4,
             "maxpool_backward: output gradient shape error");

    int batch_size = input_grad->get_shape()[0];
    int in_feats = input_grad->get_shape()[1];

    CHECK_EQ(output_grad->get_shape()[0], batch_size,
             "maxpool_backward: batch size error");
    CHECK_EQ(output_grad->get_shape()[1], in_feats,
             "maxpool_backward: batch size error");

    CHECK_EQ(indices->get_vec().size(), output_grad->get_vec().size(),
             "maxpool_backward: size mismatch between indices and output grad");

    int in_h = input_grad->get_shape()[2];
    int in_w = input_grad->get_shape()[3];
    int in_stride = in_h * in_w; // is also in_stride

    int out_h = output_grad->get_shape()[2];
    int out_w = output_grad->get_shape()[3];
    int out_stride = out_h * out_w;

    float *input_grad_raw = RAW_PTR(input_grad->get_vec());
    const float *output_grad_raw = RAW_PTR(output_grad->get_vec());
    const float *indices_raw = RAW_PTR(indices->get_vec());

    int size = batch_size * in_feats * in_h * in_w;
    int grid_size = ceil((float)size / BLOCK_SIZE);

    maxpool_backward_kernel<<<grid_size, BLOCK_SIZE>>>(
        size, input_grad_raw, output_grad_raw, indices_raw, in_h, in_w, pad_h,
        pad_w, filter_h, filter_w, stride_h, stride_w, out_h, out_w, in_stride,
        out_stride);
    CUDA_POST_KERNEL_CHECK;
}

MaxPool2D::MaxPool2D(int pad_h, int pad_w, int kernel_h, int kernel_w,
                     int stride_h, int stride_w)
    : pad_h(pad_h), pad_w(pad_w), kernel_h(kernel_h), kernel_w(kernel_w),
      stride_h(stride_h), stride_w(stride_w) {}

void MaxPool2D::forward() {
    const Array *input = prev->get_output();

    int batch_size = input->get_shape()[0];
    int in_feats = input->get_shape()[1];
    int in_h = input->get_shape()[2];
    int in_w = input->get_shape()[3];

    int out_h = (in_h + 2 * pad_h - kernel_h) / stride_h + 1;
    int out_w = (in_w + 2 * pad_w - kernel_w) / stride_w + 1;

    set_array_ptr(output, {batch_size, in_feats, out_h, out_w});
    set_array_ptr(indices, output->get_shape());

    maxpool_forward(output.get(), input, indices.get(), pad_h, pad_w, kernel_h,
                    kernel_w, stride_h, stride_w);
}

void MaxPool2D::backward() {
    const Array *input = prev->get_output();
    const Array *output_grad = next->get_grad();
    set_array_ptr(grad, input->get_shape());
    maxpool_backward(grad.get(), output_grad, indices.get(), pad_h, pad_w,
                     kernel_h, kernel_w, stride_h, stride_w);
}

} // namespace nnv2