#include "hip/hip_runtime.h"
#include "common.cuh"
#include "network.cuh"

#include <algorithm>
#include <cfloat>
#include <iostream>
#include <utility>

#include <thrust/reduce.h>

namespace nnv2 {

// Appends a layer to the network
void Network::add(Layer *layer) {
    std::unique_ptr<Layer> next(layer);
    layers.push_back(std::move(next));
}

// Introduces DataLoader, Loss and Optimizer and connect them to the layers
void Network::init(DataLoader *loader_, Loss *loss_, Optimizer *optimizer_) {
    loader = loader_;
    loss = loss_;
    optimizer = optimizer_;

    CHECK_COND(layers.size() > 0, "No layers found in the network");

    // Connect loader to the first layer
    loader->connect(layers.front().get());

    // Connect each layer to the subsequent one
    for (int i = 1; i < layers.size(); i++) {
        layers[i - 1]->connect(layers[i].get());
    }

    // Connect the last layer to loss layer
    layers.back()->connect(loss);

    // Register parameters to the optimizer
    for (int i = 0; i < layers.size(); i++) {
        optimizer->add_parameters(layers[i]->get_parameters());
    }
}

void Network::train(int epochs, bool shuffle) {
    for (int e = 0; e < epochs; e++) {
        std::cout << "[Epoch: " << e + 1 << "/" << epochs << "] ";
        loader->reset(shuffle);
        train_epoch();
        test();
    }
}

void Network::train_epoch() {
    float loss_sum = 0.0;
    int batch_count = 0;

    while (loader->has_next_train_batch()) {
        batch_count++;
        loader->load_train_batch();

        // Perform forward phase to calculate prediction
        for (int i = 0; i < layers.size(); i++) {
            layers[i]->forward();
        }

        // Calculate loss value of prediction compared to actual result
        loss_sum += loss->calculate_loss(loader->get_labels());

        // Perform backward phase to propagate the loss gradient to parameters
        // in all layers
        loss->backward();
        for (int i = layers.size() - 1; i >= 0; i--) {
            layers[i]->backward();
        }

        // Update the parameters using the gradients calculated in the backward
        // phase
        optimizer->update_parameters();
    }

    std::cout << "Avg loss (train): " << loss_sum / batch_count << "; ";
}

void Network::test() {
    float loss_sum = 0.0;
    int batch_count = 0;
    int accurate_count = 0;
    int sample_count = 0;

    while (loader->has_next_test_batch()) {
        batch_count++;
        loader->load_test_batch();

        // Perform forward phase to calculate prediction
        for (int i = 0; i < layers.size(); i++) {
            layers[i]->forward();
        }

        // Calculate loss & accuracy of prediction compared to actual result
        loss_sum += loss->calculate_loss(loader->get_labels());

        std::pair<int, int> accuracy =
            top1_accuracy(layers.back()->get_output(), loader->get_labels());
        accurate_count += accuracy.first;
        sample_count += accuracy.second;
    }

    // Print some stats here
    std::cout << "Avg loss (test): " << loss_sum / batch_count << ", ";
    std::cout << "Avg accuracy (test): " << 1.0 * accurate_count / sample_count
              << std::endl;
}

// Determines the accuracy of the batch by comparing the labels with the highest
// probability in the output batch and the actual labels.
// Returns number of accurate labels and total number of labels in a batch.
// TODO: optimize max reduce op in the kernel (and other similar reduce ops)
__global__ void top1_accuracy_kernel(int size, int *is_accurate,
                                     const float *preds, const float *y,
                                     int label_stride) {
    CUDA_GRID_STRIDE_LOOP(idx, size) {
        preds += idx * label_stride;
        y += idx * label_stride;

        float max_val = -FLT_MAX;
        int pred_label = -1;
        int y_label = -1;

        // Find label with the highest probability in the output
        for (int i = 0; i < label_stride; i++) {
            if (max_val < preds[i]) {
                max_val = preds[i];
                pred_label = i;
            }
        }
        // Find actual label
        for (int i = 0; i < label_stride; i++) {
            if (y[i] == 1) {
                y_label = i;
                break;
            }
        }
        is_accurate[idx] = (pred_label == y_label ? 1 : 0);
    }
}

std::pair<int, int> Network::top1_accuracy(const Array *preds, const Array *y) {
    int batch_size = preds->get_shape()[0];
    int label_stride = preds->get_shape()[1];

    int grid_size = utils::quotient_ceil(batch_size, BLOCK_SIZE);

    is_accurate.resize(batch_size);
    int *is_accurate_raw = RAW_PTR(is_accurate);
    const float *preds_raw = RAW_PTR(preds->get_vec());
    const float *y_raw = RAW_PTR(y->get_vec());

    top1_accuracy_kernel<<<grid_size, BLOCK_SIZE>>>(
        batch_size, is_accurate_raw, preds_raw, y_raw, label_stride);
    CUDA_POST_KERNEL_CHECK;

    int count = thrust::reduce(is_accurate.begin(), is_accurate.end());
    return std::make_pair(count, batch_size);
}

} // namespace nnv2