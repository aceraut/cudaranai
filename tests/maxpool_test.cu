#include "hip/hip_runtime.h"
#include "common.cuh"
#include "maxpool.cuh"
#include "test_utils.cuh"

#include <iostream>
#include <vector>

using namespace nnv2;

void test_maxpool_forward() {
    int batch_size = 2;
    int in_feats = 2;
    int in_h = 4;
    int in_w = 4;

    // test max pooling with no padding
    Array input({2, 2, 4, 4}, {1, 3, 2, 1, 4, 6, 5, 1, 1, 2, 1, 3, 0, 2, 4, 1,
                               0, 1, 1, 0, 1, 0, 2, 1, 2, 3, 1, 2, 1, 0, 1, 3,
                               2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2,
                               5, 4, 3, 2, 4, 5, 6, 1, 3, 6, 5, 4, 2, 1, 4, 3});

    Array output({2, 2, 2, 2});
    Array indices({2, 2, 2, 2});

    maxpool_forward(&output, &input, &indices, 0, 0, 2, 2, 2, 2);
    check_equal_vecs(output.get_vec(),
                     {6, 5, 2, 4, 1, 2, 3, 3, 2, 2, 2, 2, 5, 6, 6, 5});

    // main test
    int pad_h = 1;
    int pad_w = 1;
    int kernel_h = 2;
    int kernel_w = 2;
    int stride_h = 2;
    int stride_w = 2;

    int out_h = (in_h + 2 * pad_h - kernel_h) / stride_h + 1;
    int out_w = (in_w + 2 * pad_w - kernel_w) / stride_w + 1;

    output.resize({batch_size, in_feats, out_h, out_w});
    indices.resize({batch_size, in_feats, out_h, out_w});

    maxpool_forward(&output, &input, &indices, pad_h, pad_w, kernel_h, kernel_w,
                    stride_h, stride_w);
    check_equal_vecs(output.get_vec(),
                     {1, 3, 1, 4, 6, 3, 0, 4, 1, 0, 1, 0, 2, 3, 2, 1, 1, 3,
                      2, 2, 2, 2, 2, 2, 2, 2, 2, 5, 4, 2, 4, 6, 4, 2, 4, 3});

    std::cout << "test_maxpool_forward: Passed" << std::endl;
}

void test_maxpool_backward() {
    int batch_size = 2;
    int in_feats = 2;
    int in_h = 4;
    int in_w = 4;

    int pad_h = 1;
    int pad_w = 1;
    int kernel_h = 2;
    int kernel_w = 2;
    int stride_h = 2;
    int stride_w = 2;

    int out_h = (in_h + 2 * pad_h - kernel_h) / stride_h + 1;
    int out_w = (in_w + 2 * pad_w - kernel_w) / stride_w + 1;

    Array input({2, 2, 4, 4}, {1, 3, 2, 1, 4, 6, 5, 1, 1, 2, 1, 3, 0, 2, 4, 1,
                               0, 1, 1, 0, 1, 0, 2, 1, 2, 3, 1, 2, 1, 0, 1, 3,
                               2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2,
                               5, 4, 3, 2, 4, 5, 6, 1, 3, 6, 5, 4, 2, 1, 4, 3});

    Array output({2, 2, 3, 3});
    Array indices({2, 2, 3, 3});

    Array input_grad({2, 2, 4, 4});
    Array output_grad({2, 2, 3, 3},
                      {1, 2, 3, 4, 5, 6, 7, 8, 9, 1, 2, 3, 4, 5, 6, 7, 8, 9,
                       1, 2, 3, 4, 5, 6, 7, 8, 9, 1, 2, 3, 4, 5, 6, 7, 8, 9});

    maxpool_forward(&output, &input, &indices, 1, 1, 2, 2, 2, 2);
    maxpool_backward(&input_grad, &output_grad, &indices, 1, 1, 2, 2, 2, 2);
    check_equal_vecs(input_grad.get_vec(),
                     {1, 2, 0, 3, 4, 5, 0, 0, 0, 0, 0, 6, 7, 0, 8, 9,
                      1, 2, 0, 3, 0, 0, 0, 0, 4, 5, 0, 6, 7, 0, 8, 9,
                      1, 2, 0, 3, 4, 5, 0, 6, 0, 0, 0, 0, 7, 8, 0, 9,
                      1, 2, 0, 3, 4, 0, 5, 0, 0, 0, 0, 6, 7, 0, 8, 9});
    std::cout << "test_maxpool_back: Passed" << std::endl;
}

int main() {
    test_maxpool_forward();
    test_maxpool_backward();
}